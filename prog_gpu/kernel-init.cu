#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "tools.h"

#include <time.h>

#define N 172032 //131072
#define M 672
#define THRESHOLD 100000
#define GSIZE 1000
#define PNTSIZE 300
#define AUTO_END 10
#define ORTHO 80000
#define RADIUS 1500
#define RADSCALE 1000000000
#define LINE_SIZE 0.3

#define NCOMPASS 360

#define d2r(deg) (deg * PI / 180.0)
#define kill(s) (s->dead = true)

#define PI 3.14159265358979323846

#define MAPX 1273623389 
#define MAPY 363706170

#define True 1
#define False 0

#define DELIM "\t"

int nnum, bnum, fnum;
int point_mode = 0;

int width = 800, height = 800;
signal sig[N];
my_t compass[NCOMPASS];
int count[NCOMPASS];
int selection_mode = 0; //generator: 0, detector: 1

clock_t total_testing_time = 0;

node *Nodes;
polygon *Buildings;
polygon *Forests;

my_t mapx = 0; //1273623389;
my_t mapy = 0; //363706170;

line ga;
int toggle[10];

void load_file();
void clean_up();
void initialize();

__global__ void signal_calculation(signal *signal_list,
	const node *node_list, const polygon *building_list, const polygon *forest_list, const line *gtoa) {
	my_t gx = gtoa->x1;
	my_t gy = gtoa->y1;
	my_t ax = gtoa->x2;
	my_t ay = gtoa->y2;
	my_t zx, zy;
	int i = threadIdx.x + (blockIdx.x * blockDim.x);

	my_t px, py, test, tdist = 0, kdist = 0;
	signal sigref, sigblk;
	bool possible;

	signal *si = &signal_list[i];

	int autoend = -1;
	while (!si->dead && ++autoend < AUTO_END) {
		si->d = si->vy*si->x - si->vx*si->y;

		// case of detection
		possible = false;
		my_t d = (-si->vy*ax + si->vx*ay + si->d) / RADSCALE;
		if (-RADIUS <= d && d <= RADIUS) {
			if (si->vx != 0) {
				px = ax + (d*si->vy / RADSCALE);
				test = (px - si->x) ^ si->vx;
			}
			else {
				py = ay - (d*si->vx / RADSCALE);
				test = (py - si->y) ^ si->vy;
			}

			if (test > 0) {
				possible = true;
				zx = (si->x - ax);
				zy = (si->y - ay);
				tdist = zx*zx + zy*zy;
			}
		}


		// reflection test
		int n1, n2;
		int j, k;
		my_t test, kdist;
		my_t lx1, ly1, lx2, ly2;
		my_t Tnx, Tny, Td, pr;

		sigref.dead = true;
		int eid;

		for (j = 0; j < gtoa->bnum; j++) {
			// calculate reflection
			const polygon *p = &building_list[j];

			d = ((-si->vy)*p->x + (si->vx)*p->y + si->d) / RADSCALE;
			pr = p->radius;
			//possibly blocked if...
			if (-pr <= d && d <= pr)
			{
				for (k = 0; k < p->isize - 1; k++)
				{
					eid = 100 * i + k;
					if (si->eid == eid) continue;
					n1 = p->inodes[k];
					n2 = p->inodes[k + 1];

					lx1 = node_list[n1].x;
					ly1 = node_list[n1].y;
					lx2 = node_list[n2].x;
					ly2 = node_list[n2].y;

					Tnx = -si->vy;
					Tny = si->vx;
					Td = -(-si->vy*si->x + si->vx*si->y);
					my_t tb = Tnx*(lx2 - lx1) + Tny*(ly2 - ly1);

					if (tb == 0) { // parallel
						continue;
					}

					my_t t = -(Tnx*lx1 + Tny*ly1 + Td);
					if (t == 0 || t == tb) {
						continue;
					}
					if ((0 < t && t < tb) || (tb < t && t < 0)) {
						my_t px = lx1 + t*(lx2 - lx1) / tb;
						my_t py = ly1 + t*(ly2 - ly1) / tb;

						if (si->vx != 0) {
							test = (px - si->x) ^ si->vx;
						}
						else {
							test = (py - si->y) ^ si->vy;
						}

						if (test > 0) {
							zx = (si->x - px);
							zy = (si->y - py);
							kdist = zx*zx + zy*zy;
							if (kdist < 10) continue;
							if (sigref.dead || sigref.ss > kdist) { //if marked as alive
								my_t lnx = -(ly2 - ly1);
								my_t lny = (lx2 - lx1);
								my_t nv = lnx*si->vx + lny*si->vy;
								sigref.x = px;
								sigref.y = py;
								sigref.vx = si->vx - 2 * nv * lnx / (lnx*lnx + lny*lny);
								sigref.vy = si->vy - 2 * nv * lny / (lnx*lnx + lny*lny);
								sigref.ss = kdist;
								sigref.eid = eid;
								sigref.dead = false;
							}
						}
					}
				}
			}
		}

		// blocking test
		sigblk.dead = false;
		for (i = 0; i < gtoa->fnum; i++) {
			// calculate reflection
			const polygon *p = &forest_list[i];
			d = ((-si->vy)*p->x + (si->vx)*p->y + si->d) / RADSCALE;
			pr = p->radius;
			//possibly blocked if...
			if (-pr <= d && d <= pr)
			{
				for (k = 0; k < p->isize - 1; k++)
				{
					n1 = p->inodes[k];
					n2 = p->inodes[k + 1];

					lx1 = node_list[n1].x;
					ly1 = node_list[n1].y;
					lx2 = node_list[n2].x;
					ly2 = node_list[n2].y;

					Tnx = -si->vy;
					Tny = si->vx;
					Td = -(-si->vy*si->x + si->vx*si->y);//sigin->d;
														 // p' = p1 + t(p2-p1), T(dot)p' = 0
														 // t = -(T(dot)p1) / (T(dot)(p2 - p1))
					my_t tb = Tnx*(lx2 - lx1) + Tny*(ly2 - ly1);

					if (tb == 0) { // parallel
						continue;
					}

					my_t t = -(Tnx*lx1 + Tny*ly1 + Td);
					if (t == 0 || t == tb) continue;
					if ((0 < t && t < tb) || (tb < t && t < 0)) {
						my_t px = lx1 + t*(lx2 - lx1) / tb;
						my_t py = ly1 + t*(ly2 - ly1) / tb;

						if (si->vx != 0) {
							test = (px - si->x) ^ si->vx;
						}
						else {
							test = (py - si->y) ^ si->vy;
						}

						if (test > 0) {
							zx = (si->x - px);
							zy = (si->y - py);
							kdist = zx*zx + zy*zy;
							if (!sigblk.dead || sigblk.ss > kdist) { //if marked as alive
																	 //printf("kdist = %lld\n", kdist);
								sigblk.x = px;
								sigblk.y = py;
								sigblk.ss = kdist;
								sigblk.dead = true;
							}
						}
					}
				}
			}
		}

		if (!sigref.dead) {
			if (sigblk.dead) {
				if (possible && tdist < sigref.ss && tdist < sigblk.ss) {
					si->ss += sqrt((float)tdist);
					break;
				}
				if (sigref.ss < sigblk.ss) {
					sigref.ss = sqrt(float(sigref.ss));
					sigref.ss += si->ss;
					*si = sigref;
					continue;
				}
				else {
					kill(si);
					break;
				}
			}
			else {
				if (possible && tdist < sigref.ss) {
					si->ss += sqrt((float)tdist);
					break;
				}
				else {
					sigref.ss = sqrt(float(sigref.ss));
					sigref.ss += si->ss;
					*si = sigref;
					continue;
				}
			}
		}
		else {
			if (sigblk.dead) {
				if (possible && tdist < sigblk.ss) {
					si->ss += sqrt((float)tdist);
					break;
				}
				else {
					kill(si);
					break;
				}
			}
		}

		if (possible)
			si->ss += sqrt((float)tdist);
		else
			kill(si);
		break;
	}
	if (autoend == AUTO_END) {
		kill(si);
	}
}

////////////////// cuda
signal *dev_signals;
node *dev_nodes;
polygon *dev_buildings, *dev_forests;
line *dev_gtoa;

void freeCudaMemory() {
	hipFree(dev_signals);
	hipFree(dev_nodes);
	hipFree(dev_buildings);
	hipFree(dev_forests);
	hipFree(dev_gtoa);
}

hipError_t allocateCudaMemory() {
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}


	// Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_gtoa, sizeof(line));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_signals, N * sizeof(signal));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_nodes, nnum * sizeof(node));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.	
	cudaStatus = hipMemcpy(dev_nodes, Nodes, nnum * sizeof(node), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_buildings, bnum * sizeof(polygon));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(dev_buildings, Buildings, bnum * sizeof(polygon), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_forests, fnum * sizeof(polygon));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_forests, Forests, fnum * sizeof(polygon), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


Error:
	return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t signalCalcWithCuda()
{
	clock_t tic = clock();
	hipError_t cudaStatus;

	long double r;

	for (int i = 0; i < N; i++) {
		signal *si = &sig[i];
		r = d2r(360.0 * i / (long double)N);
		si->x = ga.x1;
		si->y = ga.y1;
		si->vx = cosl(r) * RADSCALE;
		si->vy = sinl(r) * RADSCALE;
		si->ss = 0;
		si->dead = false;
		si->eid = -1;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_signals, &sig, N * sizeof(signal), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_gtoa, &ga, sizeof(line), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	signal_calculation << <M, N / M >> >(dev_signals, dev_nodes, dev_buildings, dev_forests, dev_gtoa);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "signal_calculation launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching signal_calculation!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(sig, dev_signals, N * sizeof(signal), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	clock_t toc = clock();

	total_testing_time += toc - tic;
	return cudaStatus;
}

void convertToCompass() {
	int i, sidx;
	double sum;

	for (i = 0; i < NCOMPASS; i++) {
		compass[i] = 0;
		count[i] = 0; //initialzie
	}

	int deg;
	for (i = 0; i < N; i++) {
		deg = (int)(atan2(-sig[i].vy, -sig[i].vx) * 180 / PI);
		if (deg < 0) deg += 360;
		sidx = NCOMPASS * deg / 360;

		if (!sig[i].dead) {
			compass[sidx] = 1000000000 / sig[i].ss;
			count[sidx] = 1;
		}
		//compass[hidx]
	}
	for (i = 0; i < NCOMPASS; i++) {
		if(count[i] != 0)
			compass[i] /= count[i];
	}
}

void printOutput(){
	fprintf(stdout, "[");
	int i;
	for (i = 0; i < NCOMPASS; i++) {
		if (i == NCOMPASS - 1) {
			fprintf(stdout, "%d", compass[i]);
		}
		else {
			fprintf(stdout, "%d,", compass[i]);
		}
	}

	fprintf(stdout, "]");
}

int main(int argc, char* argv[])
{
	if (argc == 1) {
		fprintf(stderr, "usage: ./prog mapx mapy x1 y1 x2 y2");
		return -1;
	}
	int x, y;
	my_t x1, y1, x2, y2;
	mapx = atol(argv[1]);
	mapy = atol(argv[2]);

	x = atoi(argv[3]);
	y = atoi(argv[4]);
	y = height - y - 1;
	x1 = 2 * (x - width*0.5) / width * ORTHO;
	y1 = 2 * (y - height*0.5) / height * ORTHO;


	x = atoi(argv[5]);
	y = atoi(argv[6]);
	y = height - y - 1;
	x2 = 2 * (x - width*0.5) / width * ORTHO;
	y2 = 2 * (y - height*0.5) / height * ORTHO;

	initialize();
	ga.bnum = bnum;
	ga.fnum = fnum;
	printf("[");

	ga.x1 = x1;
	ga.y1 = y1;
	ga.x2 = x2;
	ga.y2 = y2;
	signalCalcWithCuda();
	convertToCompass();
	printOutput();
	printf(",");

	ga.x1 = x2;
	ga.y1 = y2;
	ga.x2 = x1;
	ga.y2 = y1;
	signalCalcWithCuda();
	convertToCompass();
	printOutput();
	printf("]");


	clean_up();

	return 0;
}

void initialize() {
	load_file();
	allocateCudaMemory();
}

void load_file() {
	int i, count;
	FILE * fp;
	char stmp[255];
	char *pstr;
	char *token;
	char *next_ptr;
	char *c;
	int nidx, bidx, fidx;

	int firstline = True;
	int isname = True;
	int ti;
	int tokidx;
	my_t mxx, mxy, mix, miy;

	fp = fopen("gamemap.txt", "rt");
	if (fp != NULL)
	{
		nidx = bidx = fidx = 0;
		fscanf(fp, "i\t%d\t%d\t%d\t%lld\t%lld\n", &nnum, &bnum, &fnum, &mapx, &mapy);
		Nodes = (node*)malloc(sizeof(node)*nnum);
		Buildings = (polygon*)malloc(sizeof(polygon)*bnum);
		Forests = (polygon*)malloc(sizeof(polygon)*fnum);

		while (!feof(fp))
		{
			pstr = fgets(stmp, sizeof(stmp), fp);
			if (pstr == NULL) break;
			if (pstr[0] == 'n') {
				double lat, lon;
				sscanf(pstr, "n\t%lf\t%lf", &lat, &lon);
				Nodes[nidx].x = (my_t)(lon*1e7 - mapx);
				Nodes[nidx].y = (my_t)(lat*1e7 - mapy);
				nidx++;
			}
			if (*pstr == 'b') {
				count = 0; //except name tag
				for (c = pstr+2; *c != NULL; c++) {
					if (*c == '\t') count++;
				}

				//Buildings[bidx].inodes = (int*)malloc(sizeof(int)*count);
				Buildings[bidx].isize = count;
				mxx = mxy = -99999;
				mix = miy = 99999;

				tokidx = 0;
				isname = True;

				/* get the first token */
				token = strtok(pstr + 2, DELIM);

				/* walk through other tokens */
				while( token != NULL )
				{
					if (isname) {
						isname = False;
						token = strtok(NULL, DELIM);
						continue;
					}
					sscanf(token, "%d", &ti);

					Buildings[bidx].inodes[tokidx] = ti;
					if (mxx < Nodes[ti].x)
						mxx = Nodes[ti].x;
					if (mxy < Nodes[ti].y)
						mxy = Nodes[ti].y;
					if (mix > Nodes[ti].x)
						mix = Nodes[ti].x;
					if (miy > Nodes[ti].y)
						miy = Nodes[ti].y;

					token = strtok(NULL, DELIM);
					tokidx++;
				}

				Buildings[bidx].x = (mxx + mix) / 2;
				Buildings[bidx].y = (mxy + miy) / 2;
				Buildings[bidx].radius = sqrtl((long double)((mxx - mix)*(mxx - mix) + (mxy - miy)*(mxy - miy))) / 2;

				bidx++;
			}
			if (*pstr == 'f') {
				count = 0;
				for (c = pstr+2; *c != NULL; c++) {
					if (*c == '\t') count++;
				}

				//Forests[fidx].inodes = (int*)malloc(sizeof(int)*count);
				Forests[fidx].isize = count;
				mxx = mxy = -99999;
				mix = miy = 99999;

				tokidx = 0;
				isname = True;

				/* get the first token */
				token = strtok(pstr + 2, DELIM);

				/* walk through other tokens */
				while( token != NULL )
				{
					if (isname) {
						isname = False;
						token = strtok(NULL, DELIM);
						continue;
					}
					sscanf(token, "%d", &ti);

					Forests[fidx].inodes[tokidx] = ti;
					if (mxx < Nodes[ti].x)
						mxx = Nodes[ti].x;
					if (mxy < Nodes[ti].y)
						mxy = Nodes[ti].y;
					if (mix > Nodes[ti].x)
						mix = Nodes[ti].x;
					if (miy > Nodes[ti].y)
						miy = Nodes[ti].y;

					token = strtok(NULL, DELIM);
					tokidx++;
				}

				Forests[fidx].x = (mxx + mix) / 2;
				Forests[fidx].y = (mxy + miy) / 2;
				Forests[fidx].radius = sqrtl((long double)((mxx - mix)*(mxx - mix) + (mxy - miy)*(mxy - miy))) / 2;

				fidx++;
			}
		}
		fclose(fp);
	}
	else
	{
		//fprintf(stderr, "File closed\n");
		//file not exist
	}
}

void clean_up() {
	int i;
	free(Nodes);
	free(Buildings);
	free(Forests);

	freeCudaMemory();
}
